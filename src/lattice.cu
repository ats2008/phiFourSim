#include "hip/hip_runtime.h"
#include "lattice.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


__global__ void testrun()
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	printf("tid : %d \n",tid);
	return;
}

__global__ void printLattice(float * lattice,uint16_t l1,uint16_t lTotal)
{
	int idx=blockIdx.x*l1+threadIdx.x;
	if(idx<lTotal)
	{
		printf("blockIdx.x*l1 + threadIdx.x = %d*%d+%d = %d -> %f \n",
					blockIdx.x,l1,threadIdx.x,blockIdx.x*l1+threadIdx.x,lattice[idx]);
	}
	return;
}

void phiFourLattice::phiFourLatticeGPUConstructor()
{
	hipMalloc(&CurrentObservablesGPU,latticeSize_);
	hipMalloc(&CurrentStateGPU,latticeSize_);
}
void phiFourLattice::phiFourLatticeGPUDistructor()
{
	hipFree(CurrentStateGPU);
	hipFree(CurrentObservablesGPU);
}

void phiFourLattice::simplePrintfFromKernel()
{

	printf("haha in the wraper \n");
	testrun<<<5,2>>>();
	printf("haha leaving the wraper \n");

	hipDeviceSynchronize();
}
void phiFourLattice::initializeLatticeGPU()
{
	return;
}

void phiFourLattice::copyStateInCPUtoGPU()
{
	std::cout<<"\n Copying to the GPU \n";
	hipMemcpy(CurrentStateGPU,CurrentStateCPU,latticeSize_,hipMemcpyHostToDevice); 	
}
void phiFourLattice::copyStateToGPUtoCPU()
{
	hipMemcpy(CurrentStateGPU,CurrentStateCPU,latticeSize_,hipMemcpyDeviceToHost); 	
}
void phiFourLattice::copyObservalblesInGPUToaCPU()
{
	hipMemcpy(CurrentObservablesCPU,CurrentObservablesGPU,5,hipMemcpyHostToDevice); 	
}
void phiFourLattice::copyObservalblesInCPUToGPU()
{
	hipMemcpy(CurrentObservablesGPU,CurrentObservablesCPU,5,hipMemcpyDeviceToHost); 	
}

void phiFourLattice::printLatticeOnGPU()
{
	int numberOfBlocks=latticeSize_/tStepCount_ + 1;
	int threadsPerBlock=tStepCount_;
	printLattice<<<numberOfBlocks,threadsPerBlock  >>>(CurrentStateGPU,xStepCount_,latticeSize_);
	hipDeviceSynchronize();
}


