#include "hip/hip_runtime.h"
#include "lattice.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>

__global__ void testrun()
{
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	printf("global tId : %d \n",tId);
	return;
}

__global__ void printLattice(float * lattice,uint16_t l1,uint16_t lTotal)
{
	int idx=blockIdx.x*l1+threadIdx.x;
	if(idx<lTotal)
	{
		printf("blockIdx.x*l1 + threadIdx.x = %d*%d+%d = %d -> %f \n",
					blockIdx.x,l1,threadIdx.x,blockIdx.x*l1+threadIdx.x,lattice[idx]);
	}
	return;
}

__device__ int  getNeighbour(int probeDim,int dir, int tId,int tStepCount_,int xyzblockSize)
{
	if(probeDim == 1 ) 
	{
		auto neibBlkIdx = ( threadIdx.x + dir < 0 or threadIdx.x + dir > blockDim.x -1 )? (blockIdx.x + dir + gridDim.x )%gridDim.x : blockIdx.x;
		auto neibThrIdx = ( threadIdx.x + dir + blockDim.x )%blockDim.x;
		
		auto xyzblockNumber = neibBlkIdx*gridDim.y*gridDim.z + blockIdx.y*gridDim.z + blockIdx.z;
		auto xyzPos         = ( xyzblockSize * xyzblockNumber * tStepCount_ ) 
				+ neibThrIdx*blockDim.y*blockDim.z + threadIdx.y*blockDim.z +threadIdx.z ;
		auto posIdx= tId*xyzblockSize + xyzPos ;
		
		return posIdx;
	}
	if(probeDim == 2 ) 
	{
		auto neibBlkIdx = ( threadIdx.y +dir < 0 or threadIdx.y + dir > blockDim.y -1 )? (blockIdx.y + dir + gridDim.y )%gridDim.y : blockIdx.y;
		auto neibThrIdx = ( threadIdx.y+ dir + blockDim.y )%blockDim.y;
		
		auto xyzblockNumber = blockIdx.x*gridDim.y*gridDim.z + neibBlkIdx*gridDim.z + blockIdx.z;
		auto xyzPos         = ( xyzblockSize * xyzblockNumber * tStepCount_ ) 
				+ threadIdx.x*blockDim.y*blockDim.z + neibThrIdx*blockDim.z +threadIdx.z ;
		auto posIdx= tId*xyzblockSize + xyzPos ;
		
		return posIdx;
	}
	if(probeDim == 3 ) 
	{
		auto neibBlkIdx = (threadIdx.z +dir < 0 or threadIdx.z + dir > blockDim.z -1 )? (blockIdx.z + dir + gridDim.z )%gridDim.z : blockIdx.z;
		auto neibThrIdx = (threadIdx.z + dir + blockDim.z )%blockDim.z;
		
		auto xyzblockNumber = blockIdx.x*gridDim.y*gridDim.z + blockIdx.y*gridDim.z + neibBlkIdx;
		auto xyzPos         = ( xyzblockSize * xyzblockNumber * tStepCount_ ) 
				+ threadIdx.x*blockDim.y*blockDim.z + threadIdx.y*blockDim.z +neibThrIdx ;
		auto posIdx= tId*xyzblockSize + xyzPos ;
		
		return posIdx;
	}
	
	return -1;
}

__global__ void checkBoardPhiFourUpdate(float* neiblatticeArray,float* currentlatticeArray,float* destlatticeArray,float *deltaEworkspace,\\
					float m2,float lambda,int mode,float tempAssignNumber, int tStepCount_, const int NTot,float * RNG_bank, float RWidth =2.0)
{
	
	auto xyzblockSize   = blockDim.x*blockDim.y*blockDim.z;
	auto xyzblockNumber = blockIdx.x*gridDim.y*gridDim.z + blockIdx.y*gridDim.z + blockIdx.z;
	auto xyzPos         = ( xyzblockSize * xyzblockNumber * tStepCount_ ) 
				+ threadIdx.x*blockDim.y*blockDim.z + threadIdx.y*blockDim.z +threadIdx.z ;
		
	auto tId = (threadIdx.x + threadIdx.y + threadIdx.z ) % 2 ;
	
	if( mode==1 && tId==0 ) tId=1;
	else if( mode==1 && tId==1 ) tId=0;
	
	while(tId<tStepCount_)
	{
		
		auto posIdx= tId*xyzblockSize + xyzPos ;
		auto phix=currentlatticeArray[posIdx];
		
		float dPhi= RWidth*(RNG_bank[2*posIdx]-0.5);
		auto neibPlus =( (tId+1 + tStepCount_)%tStepCount_)*xyzblockSize + xyzPos  ;
		auto neibMinus=( (tId-1 + tStepCount_)%tStepCount_)*xyzblockSize + xyzPos  ;
		
		float deltaE=2*(2*phix + dPhi )*dPhi- dPhi*(neiblatticeArray[neibPlus] + neiblatticeArray[neibMinus]);
		
		neibPlus  = getNeighbour(1,  1 , tId,tStepCount_,xyzblockSize);
		neibMinus = getNeighbour(1, -1 , tId,tStepCount_,xyzblockSize);
		deltaE   += 2*(2*phix + dPhi )*dPhi- dPhi*(neiblatticeArray[neibPlus] + neiblatticeArray[neibMinus]);
		
		neibPlus  = getNeighbour(2,  1 , tId,tStepCount_,xyzblockSize);
		neibMinus = getNeighbour(2, -1 , tId,tStepCount_,xyzblockSize);
		deltaE   += 2*(2*phix + dPhi )*dPhi- dPhi*(neiblatticeArray[neibPlus] + neiblatticeArray[neibMinus]);
		
		neibPlus  = getNeighbour(3,  1 , tId,tStepCount_,xyzblockSize);
		neibMinus = getNeighbour(3, -1 , tId,tStepCount_,xyzblockSize);
		deltaE   += 2*(2*phix + dPhi )*dPhi- dPhi*(neiblatticeArray[neibPlus] + neiblatticeArray[neibMinus]);
		
		deltaE   += ( (phix+dPhi)*(phix+dPhi) -phix*phix )*m2 + lambda*((phix+dPhi)*(phix+dPhi)*(phix+dPhi)*(phix+dPhi) -phix*phix*phix*phix );

		if(deltaE<0 || ( deltaE>0 && (exp(-deltaE) > RNG_bank[2*posIdx+1])) )
		{
			destlatticeArray[posIdx]  = phix+dPhi;
			deltaEworkspace[posIdx]   = deltaE;
		//	deltaEworkspace[posIdx]   = dPhi;
		}
		else
		{
			destlatticeArray[posIdx]  = phix;
			deltaEworkspace[posIdx]   = 0.0;
		
		}

		//printf(" posIdx : %d , dE : %f ,dPhi : %f, phiOld : %f , phiFinal : %f , exp(-dE) : %f , rnd : %f \n",\\
				posIdx,deltaE,dPhi,currentlatticeArray[posIdx],destlatticeArray[posIdx] ,exp(-deltaE),RNG_bank[posIdx+1]);
	 	tId+=2; 	
	}
}


__global__ void checkBoardUpdate(float* latticeArray,int mode,float tempAssignNumber, int tStepCount_, const int NTot,float * RNG_bank)
{
	
	auto xyzblockSize   = blockDim.x*blockDim.y*blockDim.z;
	auto xyzblockNumber = blockIdx.x*gridDim.y*gridDim.z + blockIdx.y*gridDim.z + blockIdx.z;
	auto xyzPos         = ( xyzblockSize * xyzblockNumber * tStepCount_ ) 
				+ threadIdx.x*blockDim.y*blockDim.z + threadIdx.y*blockDim.z +threadIdx.z ;
		
	//auto gridOffset   = gridDim.x*blockDim.x * gridDim.y*blockDim.y * gridDim.z*blockDim.z;
	auto tId = (threadIdx.x + threadIdx.y + threadIdx.z ) % 2 ;
	
	if( mode==1 && tId==0 ) tId=1;
	else if( mode==1 && tId==1 ) tId=0;
	
	// if(mode==0 && tId==0 ) tId=0;
	// if(mode==0 && tId==1 ) tId=1;
	
	while(tId<tStepCount_)
	{
		
		auto posIdx= tId*xyzblockSize + xyzPos ;
		latticeArray[posIdx]=RNG_bank[posIdx];
		
		auto neib=( (tId+1 + tStepCount_)%tStepCount_)*xyzblockSize + xyzPos  ;
		//printf("neib : %f , ",neib);
		latticeArray[neib]=-1*tempAssignNumber;
		auto neibA = getNeighbour(1, 1 , tId,tStepCount_,xyzblockSize);
		latticeArray[neibA] = -1*tempAssignNumber;
		auto neibB = getNeighbour(2, 1 , tId,tStepCount_,xyzblockSize);
		latticeArray[neibB] = -1*tempAssignNumber;
		auto neibC = getNeighbour(3, 1 , tId,tStepCount_,xyzblockSize);
		latticeArray[neibC] = -1*tempAssignNumber;
	
		printf(" xyzblockSize = %d, posIdx =%d, tID = %d ,xyzblockNumber = %d ,xyzPos = %d [ tIdX %d, tIdY %d, tIdZ %d , bidX:%d, bidY:%d, bidZ:%d ] latticeArray[%d] -> %f ( neibs : %d %d %d %d -> %f %f %f %f ) \n ",\\
			xyzblockSize,posIdx,tId,xyzblockNumber,xyzPos,threadIdx.x,threadIdx.y,threadIdx.z,\\
			blockIdx.x,blockIdx.y,blockIdx.z,posIdx,latticeArray[posIdx],\\
			neib,neibA,neibB,neibC,latticeArray[neib],latticeArray[neibA],latticeArray[neibB],latticeArray[neibC]);
		
		//assert(posIdx < NTot );
		
	 	tId+=2; 	
	}
}

__global__ void init_RNG(hiprandState* RNG_State,int tStepCount_,int arraySize)
{
	auto xyzblockSize   = blockDim.x*blockDim.y*blockDim.z;
	auto xyzblockNumber = blockIdx.x*gridDim.y*gridDim.z + blockIdx.y*gridDim.z + blockIdx.z;
	auto xyzPos         = ( xyzblockSize * xyzblockNumber * tStepCount_ ) 
				+ threadIdx.x*blockDim.y*blockDim.z + threadIdx.y*blockDim.z +threadIdx.z ;

	for(int tId=0;tId<tStepCount_;tId++)
	{
		auto posIdx= tId*xyzblockSize + xyzPos ;
		if(posIdx< arraySize)
			hiprand_init(1337,posIdx,0,RNG_State+posIdx);

	}
}

__global__ void make_rand(hiprandState* RNG_State ,float*randArray,int tStepCount_,int buffSize,int latticeSize_ ,int arraySize)
{
	auto xyzblockSize   = blockDim.x*blockDim.y*blockDim.z;
	auto xyzblockNumber = blockIdx.x*gridDim.y*gridDim.z + blockIdx.y*gridDim.z + blockIdx.z;
	auto xyzPos         = ( xyzblockSize * xyzblockNumber * tStepCount_ ) 
				+ threadIdx.x*blockDim.y*blockDim.z + threadIdx.y*blockDim.z +threadIdx.z ;
	for(int buffId=0;buffId<buffSize;buffId++)
	for(int tId=0;tId<tStepCount_;tId++)
	{
		auto xyztPosIdx = (tId*xyzblockSize + xyzPos);
		auto posIdx= buffId*latticeSize_+ xyztPosIdx ;
		if(posIdx< arraySize)
		{
			randArray[2*posIdx]=hiprand_uniform(&RNG_State[xyztPosIdx]);
			randArray[2*posIdx+1]=hiprand_uniform(&RNG_State[xyztPosIdx]);
		}
	}
}

void phiFourLattice::phiFourLatticeGPUConstructor()
{
	auto err = hipMalloc(&ObservablesBufferGPU,bufferSize*obsevablesCount*sizeof(float));
	hipDeviceSynchronize();
	if(err)
		cout<<hipGetErrorName(err)<<" : "<<hipGetErrorString(err)<<"\n";
	
	err=hipMalloc(&StatesBufferGPU,bufferSize*latticeSize_*sizeof(float));
	if(err) 	cout<<hipGetErrorName(err)<<" : "<<hipGetErrorString(err)<<"\n";
	
	err=hipMalloc(&gpuDeltaEworkspace,latticeSize_*sizeof(float));
	if(err) 	cout<<hipGetErrorName(err)<<" : "<<hipGetErrorString(err)<<"\n";


	CurrentStateGPU = StatesBufferGPU;
	CurrentObservablesGPU = ObservablesBufferGPU;
	
	dim3 blockSize(blockLen_,blockLen_,blockLen_);
	dim3 gridSize(gridLen_,gridLen_,gridLen_);

	auto RNG_bankSize = maxStepCountForSingleRandomNumberFill*latticeSize_*2;
	err=hipMalloc(&RNG_State,latticeSize_*sizeof(hiprandState));
	if(err)		cout<<hipGetErrorName(err)<<" : "<<hipGetErrorString(err)<<"\n";

	err=hipMalloc(&gpuUniforRealRandomBank,RNG_bankSize*sizeof(float));
	if(err)		cout<<hipGetErrorName(err)<<"@ cuMalloc gpuUniforRealRandomBank : "<<hipGetErrorString(err)<<"\n";
	
	cout<<" Initializing random states with "<<latticeSize_<<" seeds ";
	init_RNG<<<gridSize , blockSize >>>(RNG_State,tStepCount_,latticeSize_);
	err=hipGetLastError();
	if(err) cout<<hipGetErrorName(err)<<"@ init_RNG : "<<hipGetErrorString(err)<<"\n";
	
	fillGPURandomNumberBank();
	
	auto memsize=bufferSize*latticeSize_*sizeof(float);
	cout<<" Allocated "<<bufferSize*latticeSize_*sizeof(float)/1024.0/1024.0<<" MB of DEVICE Memory for lattice ( buffer size :  "<<bufferSize<<" ) \n";
	memsize+=bufferSize*obsevablesCount*sizeof(float);
	cout<<" Allocated "<<bufferSize*obsevablesCount*sizeof(float)/1024.0/1024.0<<" MB of DEVICE Memory for  obsevables ( buffer size :  "<<bufferSize<<" @ "<<obsevablesCount<<" ) \n";
	memsize+=latticeSize_*sizeof(float);
	cout<<" Allocated "<<latticeSize_*sizeof(float)/1024.0/1024.0<<" MB of DEVICE Memory for  gpuDeltaEworkspace \n";
	memsize+=latticeSize_*sizeof(hiprandState);
	cout<<" Allocated "<<latticeSize_*sizeof(hiprandState)/1024.0/1024.0<<" MB of DEVICE Memory for  RNG_State \n";
	memsize+=RNG_bankSize*sizeof(float);
	cout<<" Allocated "<<RNG_bankSize*sizeof(float)/1024.0/1024.0<<" MB of DEVICE Memory for  gpuUniforRealRandomBank \n";

	cout<<" Total Allocated Device Memory = "<<memsize/1024.0/1024.0  <<" MB \n";
}

void phiFourLattice::fillGPURandomNumberBank()
{
	auto RNG_bankSize = maxStepCountForSingleRandomNumberFill*latticeSize_;
	dim3 blockSize(blockLen_,blockLen_,blockLen_);
	dim3 gridSize(gridLen_,gridLen_,gridLen_);
	cout<<"Filling gpuUniforRealRandomBank with "<<RNG_bankSize<<"new random numbers\n";
	auto RNG_bufferStrides=maxStepCountForSingleRandomNumberFill;
	make_rand<<<gridSize , blockSize >>>(RNG_State,gpuUniforRealRandomBank,tStepCount_,RNG_bufferStrides,latticeSize_,RNG_bankSize);
	auto err=hipGetLastError();
	if(err)		cout<<hipGetErrorName(err)<<"@ make_rand : "<<hipGetErrorString(err)<<"\n";
	hipDeviceSynchronize();

}

void phiFourLattice::phiFourLatticeGPUDistructor()
{
	hipFree(gpuUniforRealRandomBank);
	hipFree(RNG_State);
	hipFree(StatesBufferGPU);
	hipFree(ObservablesBufferGPU);
	hipFree(gpuDeltaEworkspace);
}

void phiFourLattice::simplePrintfFromKernel()
{

	auto err=hipGetLastError();
	printf("checking the err before launch \n");
	if(err)
		cout<<hipGetErrorName(err)<<" before testrun : "<<hipGetErrorString(err)<<"\n";
	printf("haha in the wraper \n");
	testrun<<<5,2>>>();
	hipDeviceSynchronize();
	err=hipGetLastError();
	cout<<hipGetErrorName(err)<<" after testrun : "<<hipGetErrorString(err)<<"\n";
	printf("haha leaving the wraper \n");

}

void phiFourLattice::initializeLatticeGPU()
{
	return;
}

void phiFourLattice::copyStateInCPUtoGPU()
{
	std::cout<<"\n Copying to the GPU \n";
	hipMemcpy(CurrentStateGPU,CurrentStateCPU,latticeSize_*sizeof(float),hipMemcpyHostToDevice); 	
}
void phiFourLattice::copyStateInGPUtoCPU()
{
	cout<<"\n\n latticeSize_*sizeof(float) = "<<latticeSize_<<" * "<<sizeof(float)<<"\n\n";
	hipMemcpy(CurrentStateCPU,CurrentStateGPU,latticeSize_*sizeof(float),hipMemcpyDeviceToHost); 	
}

void phiFourLattice::copyBufferToCPU(int begi,int end)
{
	hipMemcpy(StatesBufferCPU,&StatesBufferGPU[begi],(end-begi)*latticeSize_*sizeof(float),hipMemcpyDeviceToHost); 	
	hipMemcpy(ObservablesBufferCPU,&ObservablesBufferGPU[begi],(end-begi)*obsevablesCount*sizeof(float),hipMemcpyDeviceToHost); 	

}

void phiFourLattice::copyObservalblesInGPUToaCPU()
{
	hipMemcpy(CurrentObservablesCPU,CurrentObservablesGPU,5,hipMemcpyHostToDevice); 	
}
void phiFourLattice::copyObservalblesInCPUToGPU()
{
	hipMemcpy(CurrentObservablesGPU,CurrentObservablesCPU,5,hipMemcpyDeviceToHost); 	
}

void phiFourLattice::printLatticeOnGPU()
{
	int numberOfBlocks=latticeSize_/tStepCount_ + 1;
	int threadsPerBlock=tStepCount_;
	printLattice<<<numberOfBlocks,threadsPerBlock  >>>(CurrentStateGPU,xStepCount_,latticeSize_);
	hipDeviceSynchronize();
}

void phiFourLattice::doGPUlatticeUpdates( int numUpdates,bool copyToCPU)
{
	dim3 blockSize(blockLen_,blockLen_,blockLen_);
	dim3 gridSize(gridLen_,gridLen_,gridLen_);
 
	std::cout<<"Launching the kerrnels for Lattice Size = "<<latticeSize_<<" ( t_d = "<<tStepCount_<<" x_d = "<<xStepCount_<<" & D = "<<dim_<<"\n"
		 <<" with grid size : "<<gridSize.x<<" , "<<gridSize.y<<" , "<<gridSize.z<<"\n"
		 <<" and block size : "<<blockSize.x<<" , "<<blockSize.y<<" , "<<blockSize.z<<"\n";	
	
	//thrust::device_ptr<float> cptr = thrust::device_pointer_cast(gpuDeltaEworkspace);

	thrust::device_ptr<float> thrust_ptr_ToDeltaEWplaceB = thrust::device_pointer_cast(gpuDeltaEworkspace);
	float currEnergy = thrust::reduce(thrust_ptr_ToDeltaEWplaceB,thrust_ptr_ToDeltaEWplaceB + latticeSize_  );
	
	EnergyBufferCPU[currentBufferPosCPU]= currEnergy/latticeSize_;
	
	for(int i=0;i<numUpdates;i++)
	{
	
		CurrentStateGPU 	= StatesBufferGPU + latticeSize_*currentBufferPosGPU;
		CurrentObservablesGPU	= ObservablesBufferGPU + obsevablesCount*currentBufferPosGPU;
		currentBufferPosGPU++;
		currentBufferPosCPU++;
		if(currentBufferPosCPU == bufferSize) 	
		{
			currentBufferPosCPU=0;
		}

		if(currentBufferPosGPU == bufferSize) 	
		{
			
			if( copyToCPU )
			{
				hipDeviceSynchronize();
				copyBufferToCPU(0,currentBufferPosGPU);
				hipDeviceSynchronize();
				writeBufferToFileGPULayout("blattice",0,currentBufferPosGPU);
			}
			currentBufferPosGPU=0;
		}
		
		auto neibLattice   = CurrentStateGPU;
		auto phixLattice   = CurrentStateGPU;
		auto destnLattice  = StatesBufferGPU + currentBufferPosGPU*latticeSize_;
		checkBoardPhiFourUpdate<<<gridSize,blockSize>>>( neibLattice , phixLattice , destnLattice, gpuDeltaEworkspace , \\
					 	m2Tilda_, lTilda_, 0 , 1.0,tStepCount_ ,latticeSize_, &gpuUniforRealRandomBank[2*currentStep*latticeSize_]);
		neibLattice   = destnLattice;
		phixLattice   = CurrentStateGPU;
		destnLattice  = destnLattice;

		checkBoardPhiFourUpdate<<<gridSize,blockSize>>>(  neibLattice , phixLattice , destnLattice, gpuDeltaEworkspace , \\
						m2Tilda_, lTilda_, 1 , 1.0,tStepCount_ ,latticeSize_, &gpuUniforRealRandomBank[2*currentStep*latticeSize_]);
		currEnergy += thrust::reduce(thrust_ptr_ToDeltaEWplaceB,thrust_ptr_ToDeltaEWplaceB + latticeSize_  );
		hipDeviceSynchronize();
		EnergyBufferCPU[currentBufferPosCPU] = currEnergy/latticeSize_;
		if(i%256==0)
		{	
			std::cout<<"Doing "<<i<<" th iter \n";
			cout<<"E = "<<currEnergy<< " , currentBufferPosCPU = "<<currentBufferPosCPU<<"  EnergyBufferCPU[currentBufferPosCPU]  : "<<EnergyBufferCPU[currentBufferPosCPU]<<"\n";
		}

		currentStep++;
		if(currentStep==maxStepCountForSingleRandomNumberFill)
		{
			fillGPURandomNumberBank();
			currentStep=0;
		}

	}

	if( copyToCPU )
	{
		hipDeviceSynchronize();
		copyBufferToCPU(0,currentBufferPosGPU);
		writeBufferToFileGPULayout("blattice",0,currentBufferPosGPU);
	}

	cout<<"\n\n_______________________________\n\n";
}

/*
void phiFourLattice::doGPUlatticeUpdates( int numUpdates)
{
	dim3 blockSize(blockLen_,blockLen_,blockLen_);
	dim3 gridSize(gridLen_,gridLen_,gridLen_);
 
	std::cout<<"Launching the kerrnels for Lattice Size = "<<latticeSize_<<" ( t_d = "<<tStepCount_<<" x_d = "<<xStepCount_<<" & D = "<<dim_<<"\n"
		 <<" with grid size : "<<gridSize.x<<" , "<<gridSize.y<<" , "<<gridSize.z<<"\n"
		 <<" and block size : "<<blockSize.x<<" , "<<blockSize.y<<" , "<<blockSize.z<<"\n";	
	
	//checkBoardUpdate<<<gridSize,blockSize>>>( CurrentStateGPU , 0 , 1.0,tStepCount_ ,latticeSize_, gpuUniforRealRandomBank );
	
	checkBoardPhiFourUpdate<<<gridSize,blockSize>>>( CurrentStateGPU , 0 , 1.0,tStepCount_ ,latticeSize_, &gpuUniforRealRandomBank[0]);
	hipDeviceSynchronize();
	cout<<"\n\n_______________________________\n\n";
	//checkBoardUpdate<<<gridSize,blockSize>>>( CurrentStateGPU , 1 , 2.0,tStepCount_ ,latticeSize_ );
	//hipDeviceSynchronize();
}
*/

