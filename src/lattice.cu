#include "hip/hip_runtime.h"
#include "lattice.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


__global__ void testrun()
{
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	printf("tId : %d \n",tId);
	return;
}

__global__ void printLattice(float * lattice,uint16_t l1,uint16_t lTotal)
{
	int idx=blockIdx.x*l1+threadIdx.x;
	if(idx<lTotal)
	{
		printf("blockIdx.x*l1 + threadIdx.x = %d*%d+%d = %d -> %f \n",
					blockIdx.x,l1,threadIdx.x,blockIdx.x*l1+threadIdx.x,lattice[idx]);
	}
	return;
}

__device__ int  getNeighbour(int probeDim,int dir, int tId,int tStepCount_,int xyzblockSize)
{
	if(probeDim == 1 ) 
	{
		auto neibBlkIdx = (blockIdx.x + dir < 0 or blockIdx.x + dir > gridDim.x -1 )? (blockIdx.x + dir + gridDim.x )%gridDim.x : blockIdx.x;
		auto neibThrIdx = (threadIdx.x + dir + blockDim.x )%blockDim.x;
		
		auto xyzblockNumber = neibBlkIdx*gridDim.y*gridDim.z + blockIdx.y*gridDim.z + blockIdx.z;
		auto xyzPos         = ( xyzblockSize * xyzblockNumber * tStepCount_ ) 
				+ neibThrIdx*blockDim.y*blockDim.z + threadIdx.y*blockDim.z +threadIdx.z ;
		auto posIdx= tId*xyzblockSize + xyzPos ;
		
		return posIdx;
	}
	if(probeDim == 2 ) 
	{
		auto neibBlkIdx = (blockIdx.y +dir < 0 or blockIdx.y + dir > gridDim.y -1 )? (blockIdx.y + dir + gridDim.y )%gridDim.y : blockIdx.y;
		auto neibThrIdx = (threadIdx.y+ dir + blockDim.y )%blockDim.y;
		
		auto xyzblockNumber = blockIdx.x*gridDim.y*gridDim.z + neibBlkIdx*gridDim.z + blockIdx.z;
		auto xyzPos         = ( xyzblockSize * xyzblockNumber * tStepCount_ ) 
				+ threadIdx.x*blockDim.y*blockDim.z + neibThrIdx*blockDim.z +threadIdx.z ;
		auto posIdx= tId*xyzblockSize + xyzPos ;
		
		return posIdx;
	}
	if(probeDim == 3 ) 
	{
		auto neibBlkIdx = (blockIdx.z +dir < 0 or blockIdx.z + dir > gridDim.z -1 )? (blockIdx.z + dir + gridDim.z )%gridDim.z : blockIdx.z;
		auto neibThrIdx = (threadIdx.z + dir + blockDim.z )%blockDim.z;
		
		auto xyzblockNumber = blockIdx.x*gridDim.y*gridDim.z + blockIdx.y*gridDim.z + neibBlkIdx;
		auto xyzPos         = ( xyzblockSize * xyzblockNumber * tStepCount_ ) 
				+ threadIdx.x*blockDim.y*blockDim.z + threadIdx.y*blockDim.z +neibThrIdx ;
		auto posIdx= tId*xyzblockSize + xyzPos ;
		
		return posIdx;
	}
	
	return -1;
}

__global__ void checkBoardUpdate(float* latticeArray,int mode,float tempAssignNumber, int tStepCount_, const int NTot)
{
	//assert(mode==0 or mode==1);
	int tIdX =  ( threadIdx.x) ;
	int tIdY =  ( threadIdx.y) ;
	int tIdZ =  ( threadIdx.z) ;
	
//	int  xyzPos     = tIdX *gridDim.y*blockDim.y * gridDim.z*blockDim.z + tIdY * gridDim.z*blockDim.z+tIdZ;
	
	auto xyzblockSize   = blockDim.x*blockDim.y*blockDim.z;
	auto xyzblockNumber = blockIdx.x*gridDim.y*gridDim.z + blockIdx.y*gridDim.z + blockIdx.z;
	auto xyzPos         = ( xyzblockSize * xyzblockNumber * tStepCount_ ) 
				+ threadIdx.x*blockDim.y*blockDim.z + threadIdx.y*blockDim.z +threadIdx.z ;
		
	//auto gridOffset   = gridDim.x*blockDim.x * gridDim.y*blockDim.y * gridDim.z*blockDim.z;
	auto tId = (threadIdx.x + threadIdx.y + threadIdx.z ) % 2 ;
	
	if( mode==1 && tId==0 ) tId=1;
	else if( mode==1 && tId==1 ) tId=0;
	
	// if(mode==0 && tId==0 ) tId=0;
	// if(mode==0 && tId==1 ) tId=1;
	
	while(tId<tStepCount_)
	{
		
		auto posIdx= tId*xyzblockSize + xyzPos ;
		latticeArray[posIdx]=tempAssignNumber;
		
		auto neib=( (tId+1 + tStepCount_)%tStepCount_)*xyzblockSize + xyzPos  ;
		//printf("neib : %f , ",neib);
		latticeArray[neib]=-1*tempAssignNumber;
		auto neibA = getNeighbour(1, 1 , tId,tStepCount_,xyzblockSize);
		latticeArray[neibA] = -1*tempAssignNumber;
		auto neibB = getNeighbour(2, 1 , tId,tStepCount_,xyzblockSize);
		latticeArray[neibB] = -1*tempAssignNumber;
		auto neibC = getNeighbour(3, 1 , tId,tStepCount_,xyzblockSize);
		latticeArray[neibC] = -1*tempAssignNumber;
	
		printf(" xyzblockSize = %d, posIdx =%d, tID = %d ,xyzblockNumber = %d ,xyzPos = %d [ tIdX %d, tIdY %d, tIdZ %d , bidX:%d, bidY:%d, bidZ:%d ] latticeArray[%d] -> %f ( neibs : %d %d %d %d -> %f %f %f %f ) \n ",\\
			xyzblockSize,posIdx,tId,xyzblockNumber,xyzPos,tIdX,tIdY,tIdZ,blockIdx.x,blockIdx.y,blockIdx.z,posIdx,latticeArray[posIdx],\\
			neib,neibA,neibB,neibC,latticeArray[neib],latticeArray[neibA],latticeArray[neibB],latticeArray[neibC]);
		
		//assert(posIdx < NTot );
		
	 	tId+=2; 	
	}
}

void phiFourLattice::phiFourLatticeGPUConstructor()
{
	hipMalloc(&CurrentObservablesGPU,latticeSize_);
	hipMalloc(&CurrentStateGPU,latticeSize_*sizeof(float));
	cout<<" Allocated "<<latticeSize_*sizeof(float)/1024.0<<" Kb of DEVICE Memory for lattice \n";
}
void phiFourLattice::phiFourLatticeGPUDistructor()
{
	hipFree(CurrentStateGPU);
	hipFree(CurrentObservablesGPU);
}

void phiFourLattice::simplePrintfFromKernel()
{

	printf("haha in the wraper \n");
	testrun<<<5,2>>>();
	printf("haha leaving the wraper \n");

	hipDeviceSynchronize();
}
void phiFourLattice::initializeLatticeGPU()
{
	return;
}

void phiFourLattice::copyStateInCPUtoGPU()
{
	std::cout<<"\n Copying to the GPU \n";
	hipMemcpy(CurrentStateGPU,CurrentStateCPU,latticeSize_*sizeof(float),hipMemcpyHostToDevice); 	
}
void phiFourLattice::copyStateInGPUtoCPU()
{
	cout<<"\n\n latticeSize_*sizeof(float) = "<<latticeSize_<<" * "<<sizeof(float)<<"\n\n";
	hipMemcpy(CurrentStateCPU,CurrentStateGPU,latticeSize_*sizeof(float),hipMemcpyDeviceToHost); 	
}
void phiFourLattice::copyObservalblesInGPUToaCPU()
{
	hipMemcpy(CurrentObservablesCPU,CurrentObservablesGPU,5,hipMemcpyHostToDevice); 	
}
void phiFourLattice::copyObservalblesInCPUToGPU()
{
	hipMemcpy(CurrentObservablesGPU,CurrentObservablesCPU,5,hipMemcpyDeviceToHost); 	
}

void phiFourLattice::printLatticeOnGPU()
{
	int numberOfBlocks=latticeSize_/tStepCount_ + 1;
	int threadsPerBlock=tStepCount_;
	printLattice<<<numberOfBlocks,threadsPerBlock  >>>(CurrentStateGPU,xStepCount_,latticeSize_);
	hipDeviceSynchronize();
}

void phiFourLattice::doGPUlatticeUpdates( int numUpdates)
{
	dim3 blockSize(blockLen_,blockLen_,blockLen_);
	dim3 gridSize(gridLen_,gridLen_,gridLen_);
 
	std::cout<<"Launching the kerrnels for Lattice Size = "<<latticeSize_<<" ( t_d = "<<tStepCount_<<" x_d = "<<xStepCount_<<" & D = "<<dim_<<"\n"
		 <<" with grid size : "<<gridSize.x<<" , "<<gridSize.y<<" , "<<gridSize.z<<"\n"
		 <<" and block size : "<<blockSize.x<<" , "<<blockSize.y<<" , "<<blockSize.z<<"\n";	
	checkBoardUpdate<<<gridSize,blockSize>>>( CurrentStateGPU , 0 , 1.0,tStepCount_ ,latticeSize_ );
	hipDeviceSynchronize();
	cout<<"\n\n_______________________________\n\n";
	//checkBoardUpdate<<<gridSize,blockSize>>>( CurrentStateGPU , 1 , 2.0,tStepCount_ ,latticeSize_ );
	//hipDeviceSynchronize();
}


